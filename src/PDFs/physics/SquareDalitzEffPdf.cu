#include "hip/hip_runtime.h"
#include <goofit/PDFs/physics/SquareDalitzEffPdf.h>
#include <goofit/PDFs/ParameterContainer.h>
#include <vector>


namespace GooFit {

__device__ fptype inPS(fptype m12, fptype m13, fptype mD, fptype mKS0, fptype mh1, fptype mh2) {

  if (m12 < pow(mKS0 + mh1, 2)) return 0;
  if (m12 > pow(mD - mh2, 2)) return 0;

  // Calculate energies of 1 and 3 particles in m12 rest frame. 
  fptype e1star = 0.5 * (m12 - mh1*mh1 + mKS0*mKS0) / sqrt(m12);
  fptype e3star = 0.5 * (mD*mD - m12 - mh2*mh2) / sqrt(m12);

  fptype minimum = pow(e1star + e3star, 2) - pow(sqrt(e1star*e1star - mKS0*mKS0) + sqrt(e3star*e3star - mh2*mh2), 2);
  if (m13 < minimum) return 0;
  fptype maximum = pow(e1star + e3star, 2) - pow(sqrt(e1star*e1star - mKS0*mKS0) - sqrt(e3star*e3star - mh2*mh2), 2);
  if (m13 > maximum) return 0;

  return 1;
}

__device__ fptype mprime (fptype m12, fptype m13, fptype mD, fptype mKS0, fptype mh1, fptype mh2) {
  // Helper function to calculate m'^2
  fptype m23 = mD*mD + mKS0*mKS0 + mh1*mh1 + mh2*mh2 - m12 - m13; 
  //fptype rootPi = -2.*ATAN2(-1.0,0.0); // Pi

  if (m23 < 0) return -99;
  fptype tmp = ((2.0*(sqrt(m23) - (mh1 + mh2))/(mD - mKS0 - (mh1 + mh2))) - 1.0);
  if (isnan(tmp)) tmp = -99;
  return tmp;
}

__device__ fptype thetaprime (fptype m12, fptype m13, fptype mD, fptype mKS0, fptype mh1, fptype mh2) {
  // Helper function to calculate theta'
  fptype m23 = mD*mD + mKS0*mKS0 + mh1*mh1 + mh2*mh2 - m12 - m13; 
  if (m23 < 0) return -99;

  fptype num = m23*( m12 - m13) + (mh2*mh2 - mh1*mh1)*(mD*mD - mKS0*mKS0);
  fptype denum = sqrt(((m23 - mh1*mh1 + mh2*mh2)*(m23 - mh1*mh1 + mh2*mh2) - 4*m23*mh2*mh2))*sqrt(((mD*mD - mKS0*mKS0 - m23)*(mD*mD - mKS0*mKS0 -m23) - 4*m23*mKS0*mKS0));
  fptype theta = -99 ;
  if (isnan(denum)) return -99;

  if (denum != 0.){
    theta = num/denum;
  }

  return theta;
}

__device__ fptype device_SquareDalitzEff (fptype* evt, ParameterContainer &pc) {

  // Define observables 
  int idx = pc.getObservable(0);
  int idy = pc.getObservable(1);
  int id_num = pc.getObservable(2);

  fptype x = RO_CACHE(evt[idx]);
  fptype y = RO_CACHE(evt[idy]);
  fptype evtIndex = RO_CACHE(evt[id_num]);

  // Define coefficients
  fptype c0 = pc.getParameter(0);
  fptype c1 = pc.getParameter(1);
  fptype c2 = pc.getParameter(2);
  fptype c3 = pc.getParameter(3);
  fptype c4 = pc.getParameter(4);
  fptype c5 = pc.getParameter(5);
  fptype c6 = pc.getParameter(6);

  // Define constvals
  fptype mD = pc.getParameter(7);
  fptype mKS0 = pc.getParameter(8);
  fptype mh1 = pc.getParameter(9);
  fptype mh2 = pc.getParameter(10);

  pc.incrementIndex(1, pc.getNumParameters(), pc.getNumConstants(), pc.getNumObservables(), 1);

  // Check phase space
  if (inPS == 0) return 0;
  
  // Call helper functions
  fptype thetap = thetaprime(x,y,mD,mKS0,mh1,mh2); 
  if (thetap > 1. || thetap < -1.) return 0; 

  fptype m23 = mD*mD + mKS0*mKS0 + mh1*mh1 + mh2*mh2 - x - y; 
  if (m23 < 0) return 0;

  fptype ret = c0*m23*m23 + c1*m23 + c2*m23*thetap*thetap + c3*thetap*thetap + c4*thetap + c5 + c6*m23*thetap; 
  
  return ret; 
}

__device__ device_function_ptr ptr_to_SquareDalitzEff = device_SquareDalitzEff; 

__host__ __device__ SquareDalitzEffPdf::SquareDalitzEffPdf (std::string n, 
				        std::vector<Observable> obses, 
					std::vector<Variable> coeffs, 
					std::vector<Variable> constvals) 
  : GooPdf("SquareDalitzEffPdf", n, obses, coeffs, constvals) {

  // Register observables - here m12, m13 and dtime
  for (unsigned int i = 0; i < obses.size(); ++i) {
    registerObservable(obses[i]);
  }

  // Register constvals
  for (std::vector<Variable>::iterator v = constvals.begin(); v != constvals.end(); ++v) {
    registerParameter(v);
  }

  // Register coefficients
  for (std::vector<Variable>::iterator c = coeffs.begin(); c != coeffs.end(); ++c) {
    registerParameter(c);
  }

  registerFunction("ptr_to_SquareDalitzEff", ptr_to_SquareDalitzEff);

  initialize();

}

} // namespace GooFit